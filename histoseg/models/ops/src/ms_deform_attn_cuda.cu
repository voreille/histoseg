#include "hip/hip_runtime.h"
#include <vector>
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#define CUDA_KERNEL_LOOP(i, n)                                 \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); \
       i += blockDim.x * gridDim.x)

const int CUDA_NUM_THREADS = 1024;
inline int GET_BLOCKS(const int N, const int num_threads = CUDA_NUM_THREADS) {
  return (N + num_threads - 1) / num_threads;
}

template <typename scalar_t>
__device__ scalar_t ms_deform_attn_im2col_bilinear(const scalar_t* &bottom_data, 
                                                    const int &height, const int &width, 
                                                    const int &nheads, const int &channels,
                                                    const scalar_t &h, const scalar_t &w, 
                                                    const int &m, const int &c) {
  const int h_low = floor(h);
  const int w_low = floor(w);
  const int h_high = h_low + 1;
  const int w_high = w_low + 1;

  const scalar_t lh = h - h_low;
  const scalar_t lw = w - w_low;
  const scalar_t hh = 1 - lh, hw = 1 - lw;

  const int w_stride = nheads * channels;
  const int h_stride = width * w_stride;
  const int h_low_ptr_offset = h_low * h_stride;
  const int h_high_ptr_offset = h_low_ptr_offset + h_stride;
  const int w_low_ptr_offset = w_low * w_stride;
  const int w_high_ptr_offset = w_low_ptr_offset + w_stride;
  const int base_ptr = m * channels + c;

  scalar_t v1 = 0;
  if (h_low >= 0 && w_low >= 0) {
    const int ptr1 = h_low_ptr_offset + w_low_ptr_offset + base_ptr;
    v1 = bottom_data[ptr1];
  }
  scalar_t v2 = 0;
  if (h_low >= 0 && w_high <= width - 1) {
    const int ptr2 = h_low_ptr_offset + w_high_ptr_offset + base_ptr;
    v2 = bottom_data[ptr2];
  }
  scalar_t v3 = 0;
  if (h_high <= height - 1 && w_low >= 0) {
    const int ptr3 = h_high_ptr_offset + w_low_ptr_offset + base_ptr;
    v3 = bottom_data[ptr3];
  }
  scalar_t v4 = 0;
  if (h_high <= height - 1 && w_high <= width - 1) {
    const int ptr4 = h_high_ptr_offset + w_high_ptr_offset + base_ptr;
    v4 = bottom_data[ptr4];
  }

  const scalar_t w1 = hh * hw, w2 = hh * lw, w3 = lh * hw, w4 = lh * lw;

  const scalar_t val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);
  return val;
}

template <typename scalar_t>
__global__ void ms_deformable_im2col_gpu_kernel(const int n,
                                                const scalar_t *data_value,
                                                const int64_t *data_spatial_shapes,
                                                const int64_t *data_level_start_index,
                                                const scalar_t *data_sampling_loc,
                                                const scalar_t *data_attn_weight,
                                                const int batch_size,
                                                const int spatial_size,
                                                const int num_heads,
                                                const int channels,
                                                const int num_levels,
                                                const int num_query,
                                                const int num_point,
                                                scalar_t *data_col) {
  CUDA_KERNEL_LOOP(index, n) {
    int _temp = index;
    const int c_col = _temp % channels;
    _temp /= channels;
    const int sampling_index = _temp;
    const int m_col = _temp % num_heads;
    _temp /= num_heads;
    const int q_col = _temp % num_query;
    _temp /= num_query;
    const int b_col = _temp;

    scalar_t *data_col_ptr = data_col + index;
    int data_weight_ptr = sampling_index * num_levels * num_point;
    int data_loc_w_ptr = data_weight_ptr << 1;
    const int qid_stride = num_heads * channels;
    const int data_value_ptr_init_offset = b_col * spatial_size * qid_stride;
    scalar_t col = 0;

    for (int l_col=0; l_col < num_levels; ++l_col) {
      const int level_start_id = data_level_start_index[l_col];
      const int spatial_h_ptr = l_col << 1;
      const int spatial_h = data_spatial_shapes[spatial_h_ptr];
      const int spatial_w = data_spatial_shapes[spatial_h_ptr + 1];
      const scalar_t *data_value_ptr = data_value + data_value_ptr_init_offset + level_start_id * qid_stride;
      for (int p_col=0; p_col < num_point; ++p_col) {
        const scalar_t loc_w = data_sampling_loc[data_loc_w_ptr];
        const scalar_t loc_h = data_sampling_loc[data_loc_w_ptr + 1];
        const scalar_t weight = data_attn_weight[data_weight_ptr];

        const scalar_t h_im = loc_h * spatial_h - 0.5;
        const scalar_t w_im = loc_w * spatial_w - 0.5;

        if (h_im > -1 && w_im > -1 && h_im < spatial_h && w_im < spatial_w) {
          col += ms_deform_attn_im2col_bilinear(data_value_ptr, spatial_h, spatial_w, num_heads, channels, h_im, w_im, m_col, c_col) * weight;
        }

        data_weight_ptr += 1;
        data_loc_w_ptr += 2;
      }
    }
    *data_col_ptr = col;
  }
}

at::Tensor ms_deform_attn_cuda_forward(
    const at::Tensor &value,
    const at::Tensor &spatial_shapes,
    const at::Tensor &level_start_index,
    const at::Tensor &sampling_loc,
    const at::Tensor &attn_weight,
    const int im2col_step)
{
  AT_ASSERTM(value.is_contiguous(), "value tensor has to be contiguous");
  AT_ASSERTM(spatial_shapes.is_contiguous(), "spatial_shapes tensor has to be contiguous");
  AT_ASSERTM(level_start_index.is_contiguous(), "level_start_index tensor has to be contiguous");
  AT_ASSERTM(sampling_loc.is_contiguous(), "sampling_loc tensor has to be contiguous");
  AT_ASSERTM(attn_weight.is_contiguous(), "attn_weight tensor has to be contiguous");

  AT_ASSERTM(value.type().is_cuda(), "value must be a CUDA tensor");
  AT_ASSERTM(spatial_shapes.type().is_cuda(), "spatial_shapes must be a CUDA tensor");
  AT_ASSERTM(level_start_index.type().is_cuda(), "level_start_index must be a CUDA tensor");
  AT_ASSERTM(sampling_loc.type().is_cuda(), "sampling_loc must be a CUDA tensor");
  AT_ASSERTM(attn_weight.type().is_cuda(), "attn_weight must be a CUDA tensor");

  const int batch = value.size(0);
  const int spatial_size = value.size(1);
  const int num_heads = value.size(2);
  const int channels = value.size(3);

  const int num_levels = spatial_shapes.size(0);

  const int num_query = sampling_loc.size(1);
  const int num_point = sampling_loc.size(4);

  const int im2col_step_ = std::min(batch, im2col_step);

  AT_ASSERTM(batch % im2col_step_ == 0, "batch(%d) must divide im2col_step(%d)", batch, im2col_step_);

  auto output = at::zeros({batch, num_query, num_heads, channels}, value.options());

  const int batch_n = im2col_step_;
  auto output_n = output.view({batch/im2col_step_, batch_n, num_query, num_heads, channels});
  auto per_value_size = spatial_size * num_heads * channels;
  auto per_sample_loc_size = num_query * num_heads * num_levels * num_point * 2;
  auto per_attn_weight_size = num_query * num_heads * num_levels * num_point;
  for (int n = 0; n < batch/im2col_step_; ++n) {
    auto columns = output_n.select(0, n);
    AT_DISPATCH_FLOATING_TYPES(value.type(), "ms_deform_attn_forward_cuda", ([&] {
      ms_deformable_im2col_gpu_kernel<scalar_t>
          <<<GET_BLOCKS(batch_n * num_query * num_heads * channels), CUDA_NUM_THREADS,
             0, at::cuda::getCurrentCUDAStream()>>>(
          batch_n * num_query * num_heads * channels,
          value.data<scalar_t>() + n * im2col_step_ * per_value_size,
          spatial_shapes.data<int64_t>(),
          level_start_index.data<int64_t>(),
          sampling_loc.data<scalar_t>() + n * im2col_step_ * per_sample_loc_size,
          attn_weight.data<scalar_t>() + n * im2col_step_ * per_attn_weight_size,
          batch_n, spatial_size, num_heads, channels, num_levels, num_query, num_point,
          columns.data<scalar_t>());
    }));
  }

  AT_CUDA_CHECK(hipGetLastError());
  return output;
}

at::Tensor ms_deform_attn_cuda_backward(
    const at::Tensor &value,
    const at::Tensor &spatial_shapes,
    const at::Tensor &level_start_index,
    const at::Tensor &sampling_loc,
    const at::Tensor &attn_weight,
    const at::Tensor &grad_output,
    const int im2col_step)
{
  // Simplified backward pass - would need full implementation
  auto grad_value = at::zeros_like(value);
  auto grad_sampling_loc = at::zeros_like(sampling_loc);
  auto grad_attn_weight = at::zeros_like(attn_weight);
  
  // Return concatenated gradients
  return at::cat({grad_value.view({-1}), grad_sampling_loc.view({-1}), grad_attn_weight.view({-1})});
}
